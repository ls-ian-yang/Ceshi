
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10000000

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

//out = a + b, n is length
void vector_add(float *out, float *a, float *b, int n){
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;

    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    vector_add(out, a, b, N);
    
    for(int i = 0; i < 10; i++)
    {
        printf("%f ",out[i]);
    }
    printf("\n");
    cuda_hello<<<1,1>>>();
    return 0;
}